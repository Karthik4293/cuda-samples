#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "png_helper.h"

texture <uchar4, 2, hipReadModeElementType> tex;

#define PRIMARY_THRESHOLD 125
#define SECONDARY_THRESHOLD 75

__device__ unsigned char clamp(int n) {
	return max(0, min(255, n));
}

__device__ int sobel(int a, int b, int c, int d, int e, int f) {
	return ((a + 2*b + c) - (d + 2*e + f));
}

__global__ void sobel_kernel(uchar4 *out, int width, int height) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	
	if(x < width && y < height) {
		uchar4 x0, x1, x2, x3, x4, x5, x6, x7, x8;
		x0 = tex2D(tex, x-1, y-1);
		x1 = tex2D(tex, x, y-1);
		x2 = tex2D(tex, x+1, y);
		x3 = tex2D(tex, x-1, y);
		x4 = tex2D(tex, x, y);
		x5 = tex2D(tex, x+1, y);
		x6 = tex2D(tex, x-1, y+1);
		x7 = tex2D(tex, x, y+1);
		x8 = tex2D(tex, x+1, y+1);
		
		int dfdy_r = sobel(x6.x, x7.x, x8.x, x0.x, x1.x, x2.x);
		int dfdx_r = sobel(x2.x, x5.x, x8.x, x0.x, x3.x, x6.x);
		
		int dfdy_g = sobel(x6.y, x7.y, x8.y, x0.y, x1.y, x2.y);
		int dfdx_g = sobel(x2.y, x5.y, x8.y, x0.y, x3.y, x6.y);
		
		int dfdy_b = sobel(x6.z, x7.z, x8.z, x0.z, x1.z, x2.z);
		int dfdx_b = sobel(x2.z, x5.z, x8.z, x0.z, x3.z, x6.z);
		
		int gradient_r = abs(dfdy_r) + abs(dfdy_r);
		int gradient_g = abs(dfdy_g) + abs(dfdy_g);
		int gradient_b = abs(dfdy_b) + abs(dfdy_b);
		
		/*
		int dir_r = atanf(dfdy_r/dfdx_r);
		int dir_g = atanf(dfdy_g/dfdx_g);
		int dir_b = atanf(dfdy_b/dfdx_b);
		*/
		
		float mean_gradient = (gradient_r + gradient_g + gradient_b) / 3.0f;
		unsigned char edge = (mean_gradient > PRIMARY_THRESHOLD);
		unsigned char slight_edge = (mean_gradient > SECONDARY_THRESHOLD);
		
		uchar4 new_pixel = (uchar4) {0,0,0,255};
		
		new_pixel.x = 255 * edge | 125 * slight_edge;
		new_pixel.y = 255 * edge | 125 * slight_edge;
		new_pixel.z = 255 * edge | 125 * slight_edge;
		
		out[x + y * width] = new_pixel;
	}
}

extern "C" void sobel_wrapper(struct hipArray *in, uchar4 *out, png_t *info) {
	dim3 threads(16,16);
	dim3 blocks((info->width)/16 + 1, (info->height)/16 + 1);
	
	hipChannelFormatDesc channel_desc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
	hipBindTextureToArray(tex, in, channel_desc);
	
	tex.addressMode[0] = hipAddressModeClamp;
	tex.addressMode[1] = hipAddressModeClamp;
	tex.filterMode = hipFilterModePoint;
	tex.normalized = false;
	
	sobel_kernel<<<blocks, threads>>>(out, info->width, info->height);
	hipUnbindTexture(tex);
}