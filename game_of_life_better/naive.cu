#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "dimensions.h"
// no caching version.
__global__ void naive_game_of_life(int *current, int *future) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	
	if(x < DIM_X && y < DIM_Y) { // MUST MAKE SURE X, Y ARE GOOD! OTHERWISE HAVOC!
		int neighbor_count = 0;
		
		neighbor_count += current[y * DIM_X + ((x - 1 + DIM_X) % DIM_X)];
		neighbor_count += current[y * DIM_X + ((x + 1) % DIM_X)];
		neighbor_count += current[((y - 1 + DIM_Y) % DIM_Y) * DIM_X + x];
		neighbor_count += current[((y + 1) % DIM_Y) * DIM_X + x];
		neighbor_count += current[((y - 1 + DIM_Y) % DIM_Y) * DIM_X + ((x - 1 + DIM_X) % DIM_X)];
		neighbor_count += current[((y - 1 + DIM_Y) % DIM_Y) * DIM_X + ((x + 1) % DIM_X)];
		neighbor_count += current[((y + 1) % DIM_Y) * DIM_X + ((x - 1 + DIM_X) % DIM_X)];
		neighbor_count += current[((y + 1) % DIM_Y) * DIM_X + ((x + 1) % DIM_X)];

		if(neighbor_count == 3) {
			future[y * DIM_X + x] = 1;
		} else if(neighbor_count == 2 && current[y * DIM_X + x] == 1) {
			future[y * DIM_X + x] = 1;
		} else {
			future[y * DIM_X + x] = 0;
		}
	}
}

// the wrapper around the kernel call for main program to call.
extern "C" void naive_game_of_life_wrapper(int *current, int *future) {
	dim3 threads(16,16);
	dim3 blocks(DIM_X/16 + 1, DIM_Y/16 + 1);
	
	naive_game_of_life<<<blocks, threads>>>(current, future);
}