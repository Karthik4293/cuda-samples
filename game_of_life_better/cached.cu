#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "dimensions.h"
// no caching version.
__global__ void cached_game_of_life(int *current, int *future) {
	__shared__ int cache[18][18];
	
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	
	int local_x = threadIdx.x;
	int cache_x = local_x + 1;
	int local_y = threadIdx.y;
	int cache_y = local_y + 1;
	
	if(x < DIM_X && y < DIM_Y) { // MUST MAKE SURE X, Y ARE GOOD! OTHERWISE HAVOC!
		cache[cache_x - 1][cache_y - 1] = current[((y - 1 + DIM_Y) % DIM_Y) * DIM_X + ((x - 1 + DIM_X) % DIM_X)];
		cache[cache_x - 1][cache_y + 1] = current[((y + 1) % DIM_Y) * DIM_X + ((x - 1 + DIM_X) % DIM_X)];
		cache[cache_x + 1][cache_y - 1] = current[((y - 1 + DIM_Y) % DIM_Y) * DIM_X + ((x + 1) % DIM_X)];
		cache[cache_x + 1][cache_y + 1] = current[((y + 1) % DIM_Y) * DIM_X + ((x + 1) % DIM_X)];
		
		__syncthreads();
		
		int neighbor_count = 0;
		
		neighbor_count += cache[cache_x - 1][cache_y];
		neighbor_count += cache[cache_x + 1][cache_y];
		
		neighbor_count += cache[cache_x][cache_y - 1];
		neighbor_count += cache[cache_x][cache_y + 1];
		
		neighbor_count += cache[cache_x - 1][cache_y - 1];
		neighbor_count += cache[cache_x - 1][cache_y + 1];
		
		neighbor_count += cache[cache_x + 1][cache_y - 1];
		neighbor_count += cache[cache_x + 1][cache_y + 1];

		if(neighbor_count == 3) {
			future[y * DIM_X + x] = 1;
		} else if(neighbor_count == 2 && cache[cache_x][cache_y] == 1) {
			future[y * DIM_X + x] = 1;
		} else {
			future[y * DIM_X + x] = 0;
		}
	}
}

// the wrapper around the kernel call for main program to call.
void cached_game_of_life_wrapper(int *current, int *future) {
	dim3 threads(16,16);
	dim3 blocks(DIM_X/16 + 1, DIM_Y/16 + 1);
	
	cached_game_of_life<<<blocks, threads>>>(current, future);
}