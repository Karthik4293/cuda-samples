#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

extern "C" struct hipArray* setup_cuda_array(uchar4 *in, int width, int height) {
	hipChannelFormatDesc channel_desc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
	
	struct hipArray *array;
	hipMallocArray(&array, &channel_desc, width, height);
	hipMemcpyToArray(array, 0, 0, in, width*height*sizeof(uchar4), hipMemcpyHostToDevice);

	return array;
}